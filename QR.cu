/* 
 *  QR Factorization Dense Linear Solver
 *
 *  filename: test_cusolver_cuda6d5.cpp
 *
 *  module add cudatoolkit
 *  compile:  nvcc -o test_cusolver_cuda6d5 test_cusolver_cuda6d5.cpp -lcublas -lcusolver
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include <hip/hip_runtime.h>

#include <hipblas.h>
#include <hipsolver.h>

void printMatrix(int m, int n, const double*A, int lda, const char* name)
{
    for(int row = 0 ; row < m ; row++){
        for(int col = 0 ; col < n ; col++){
            double Areg = A[row + col*lda];
            printf("%s(%d,%d) = %f\n", name, row+1, col+1, Areg);
        }
    } 
}

int main(int argc, char*argv[])
{
    hipsolverHandle_t cudenseH = NULL;
    hipblasHandle_t cublasH = NULL;
    hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    hipError_t cudaStat4 = hipSuccess;
    const int m = 5;
    const int n = 3;
    const int lda = m;
    const int ldb = m;
    const int nrhs = 1; // number of right hand side vectors
/*       | 1 0 2 |
 *   A = | 0 3 0 | 
 *       | 4 0 0 |
 *       | 0 0 5 |
 *       | 6 0 0 |
 *   x* = ( 1 1 2 )'
 *   b = ( 5 3 4 10 6 )'
 */
    double A[lda*n] = { 
        1.0, 0.0, 4.0, 0.0, 6.0, 
        0.0, 3.0, 0.0, 0.0, 0.0,
        2.0, 0.0, 0.0, 5.0, 0.0
        };
//    double X[ldb*nrhs] = { 1.0, 1.0, 1.0}; // exact solution
    double B[ldb*nrhs] = { 5.0, 3.0, 4.0, 10.0, 6.0};
    double XC[ldb*nrhs]; // solution matrix from GPU

    double *d_A = NULL; // linear memory of GPU
    double *d_tau = NULL; // linear memory of GPU
    double *d_B  = NULL;
    int *devInfo = NULL; // info in gpu (device copy)
    double *d_work = NULL;
    int  lwork = 0;

    int info_gpu = 0;

    const double one = 1;
    printf("A = (matlab base-1)\n");
    printMatrix(m, n, A, lda, "A");
    printf("=====\n");
    printf("B = (matlab base-1)\n");
    printMatrix(m, nrhs, B, ldb, "B");
    printf("=====\n");

// step 1: create cudense/cublas handle
    cusolver_status = hipsolverDnCreate(&cudenseH);
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

    cublas_status = hipblasCreate(&cublasH);
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

// step 2: copy A and B to device
    cudaStat1 = hipMalloc ((void**)&d_A  , sizeof(double) * lda * n);
    cudaStat2 = hipMalloc ((void**)&d_tau, sizeof(double) * n);
    cudaStat3 = hipMalloc ((void**)&d_B  , sizeof(double) * ldb * nrhs);
    cudaStat4 = hipMalloc ((void**)&devInfo, sizeof(int));
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);
    cudaStat1 = hipMemcpy(d_A, A, sizeof(double) * lda * n   , hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(d_B, B, sizeof(double) * ldb * nrhs, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);

// step 3: query working space of geqrf and ormqr
    cusolver_status = hipsolverDnDgeqrf_bufferSize(
        cudenseH,
        m,
        n,
        d_A,
        lda,
        &lwork);
    assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);

    cudaStat1 = hipMalloc((void**)&d_work, sizeof(double)*lwork);
    assert(hipSuccess == cudaStat1);

// step 4: compute QR factorization
    cusolver_status = hipsolverDnDgeqrf(
        cudenseH,
        m,
        n,
        d_A,
        lda,
        d_tau,
        d_work,
        lwork,
        devInfo);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    assert(hipSuccess == cudaStat1);

    // check if QR is good or not
    cudaStat1 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);

    printf("after geqrf: info_gpu = %d\n", info_gpu);
    assert(0 == info_gpu);

// step 5: compute Q^T*B
    cusolver_status= hipsolverDnDormqr(
        cudenseH,
        HIPBLAS_SIDE_LEFT,
        HIPBLAS_OP_T,
        m,
        nrhs,
        n,
        d_A,
        lda,
        d_tau,
        d_B,
        ldb,
        d_work,
        lwork,
        devInfo);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    assert(hipSuccess == cudaStat1);

    // check if QR is good or not
    cudaStat1 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);

    printf("after ormqr: info_gpu = %d\n", info_gpu);
    assert(0 == info_gpu);


// step 6: compute x = R \ Q^T*B

    cublas_status = hipblasDtrsm(
         cublasH,
         HIPBLAS_SIDE_LEFT,
         HIPBLAS_FILL_MODE_UPPER,
         HIPBLAS_OP_N,
         HIPBLAS_DIAG_NON_UNIT,
         n,
         nrhs,
         &one,
         d_A,
         lda,
         d_B,
         ldb);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);
    assert(hipSuccess == cudaStat1);

    cudaStat1 = hipMemcpy(XC, d_B, sizeof(double)*ldb*nrhs, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);

    printf("X = (matlab base-1)\n");
    printMatrix(n, nrhs, XC, ldb, "X");

// free resources
    if (d_A    ) hipFree(d_A);
    if (d_tau  ) hipFree(d_tau);
    if (d_B    ) hipFree(d_B);
    if (devInfo) hipFree(devInfo);
    if (d_work ) hipFree(d_work);


    if (cublasH ) hipblasDestroy(cublasH);
    if (cudenseH) hipsolverDnDestroy(cudenseH);

    hipDeviceReset();

    return 0; 
}
